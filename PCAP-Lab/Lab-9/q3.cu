#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "hip/hip_runtime.h"



__device__ int power(int a, int n) {
    int res = 1;
    for (int i=0; i<n; i++)
        res *= a;
    return res;
}

__device__ int binary (int a) {
    int bin=0, n=1;
    while (a>0) {
        bin += n * (a % 2);
        a /= 2;
        n *= 10;
    }
    return bin;
}

__device__ int ones_compement (int a) {
    int x = int(log2(a)) + 1;
    return binary( a ^ (power(2, x) - 1) );
}

__global__ void changeMat (int* A, int* B) {
    int rowId = threadIdx.x, colId = threadIdx.y;
    int ha = blockDim.x, wa = blockDim.y;

    if (rowId==0 || rowId==ha-1 || colId==0 || colId==wa-1)
        B[rowId*wa+colId] = A[rowId*wa+colId];
    else {
        B[rowId*wa+colId] = ones_compement(A[rowId*wa+colId]);
    }
}


int main () {
    int ha, wa;

    printf("Enter number of rows of matrix A: ");
    scanf(" %d", &ha);
    printf("Enter number of columns of matrix A: ");
    scanf(" %d", &wa);

    int A[ha*wa], B[ha*wa], *d_A, *d_B;

    printf("Enter %d elements of matrix A:\n", ha*wa);

    for (int i=0; i<ha*wa; i++) {
        scanf(" %d", A+i);
    }


    hipMalloc((void **) &d_A, ha * wa * sizeof(int));
    hipMalloc((void **) &d_B, ha * wa * sizeof(int));

    hipMemcpy(d_A, A, ha * wa * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim = dim3(ha, wa);

    changeMat <<< 1, blockDim >>> (d_A, d_B);

    hipMemcpy(B, d_B, ha * wa * sizeof(int), hipMemcpyDeviceToHost);

    printf("Resultant vector:\n");
    for (int i=0; i<ha; i++) {
        for (int j=0; j<wa; j++) {
            printf("%d ", B[i*wa+j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
}
