#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "hip/hip_runtime.h"



__device__ int power(int a, int n) {
    int res = 1;
    for (int i=0; i<n; i++)
        res *= a;
    return res;
}

__global__ void changeMat (int* A, int* B) {
    int rowId = threadIdx.x, colId = threadIdx.y;
    int wa = blockDim.y;

    B[rowId*wa+colId] = power(A[rowId*wa+colId], rowId+1);
}


int main () {
    int ha, wa;

    printf("Enter number of rows of matrix A: ");
    scanf(" %d", &ha);
    printf("Enter number of columns of matrix A: ");
    scanf(" %d", &wa);

    int A[ha*wa], B[ha*wa], *d_A, *d_B;

    printf("Enter %d elements of matrix A:\n", ha*wa);

    for (int i=0; i<ha*wa; i++) {
        scanf(" %d", A+i);
    }


    hipMalloc((void **) &d_A, ha * wa * sizeof(int));
    hipMalloc((void **) &d_B, ha * wa * sizeof(int));

    hipMemcpy(d_A, A, ha * wa * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim = dim3(ha, wa);

    changeMat <<< 1, blockDim >>> (d_A, d_B);

    hipMemcpy(B, d_B, ha * wa * sizeof(int), hipMemcpyDeviceToHost);

    printf("Resultant vector:\n");
    for (int i=0; i<ha; i++) {
        for (int j=0; j<wa; j++) {
            printf("%d ", B[i*wa+j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
}
