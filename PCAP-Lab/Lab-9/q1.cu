#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"



__global__ void sparseMatVecMul (int* data, int* data_cols, int *rows, int* V, int* C, int n) {
    int rowId = threadIdx.x;
    int sum = 0;
    for (int i=rows[rowId]; i<rows[rowId+1]; i++) {
        sum += data[i] * V[data_cols[i]];
    }
    printf("%d ---> %d\n", rowId, sum);
    C[rowId] = sum;
}


int main () {
    int ha, wa;

    printf("Enter number of rows of matrix A: ");
    scanf(" %d", &ha);
    printf("Enter number of columns of matrix A: ");
    scanf(" %d", &wa);

    int A[ha*wa], C[ha], V[wa], *d_data, *d_data_cols, *d_rows, *d_V, *d_C, n = 0;

    printf("Enter %d elements of matrix A:\n", ha*wa);

    for (int i=0; i<ha*wa; i++) {
        scanf(" %d", A+i);
        if (A[i] != 0)
            n++;
    }

    printf("Enter %d elements of vector:\n", wa);

    for (int i=0; i<wa; i++) {
        scanf(" %d", V+i);
    }

    int data[n], data_cols[n], rows[n+1], k=0;
    rows[0] = 0;

    for (int i=0; i<ha; i++) {
        rows[i+1] = rows[i];
        for (int j=0; j<wa; j++) {
            if (A[i*wa+j] != 0) {
                data[k] = A[i*wa+j];
                data_cols[k] = j;
                rows[i+1]++;
                k++;
            }
        }
    }

    for (int i=0; i<wa*ha; i++) {
        printf("%d ", A[i]);
    }
    printf("\n");
    for (int i=0; i<n; i++) {
        printf("%d ", data[i]);
    }
    printf("\n");
    for (int i=0; i<n; i++) {
        printf("%d ", data_cols[i]);
    }
    printf("\n");
    for (int i=0; i<ha+1; i++) {
        printf("%d ", rows[i]);
    }
    printf("\n");

    hipMalloc((void **) &d_data, n * sizeof(int));
    hipMalloc((void **) &d_data_cols, n * sizeof(int));
    hipMalloc((void **) &d_rows, (ha+1) * sizeof(int));
    hipMalloc((void **) &d_V, wa * sizeof(int));
    hipMalloc((void **) &d_C, ha * sizeof(int));

    hipMemcpy(d_data, data, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_data_cols, data_cols, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rows, rows, (ha+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, wa * sizeof(int), hipMemcpyHostToDevice);

    sparseMatVecMul <<< 1, ha >>> (d_data, d_data_cols, d_rows, d_V, d_C, n);

    hipMemcpy(C, d_C, ha * sizeof(int), hipMemcpyDeviceToHost);

    printf("Resultant vector:\n");
    for (int i=0; i<ha; i++) {
        printf("%d ", C[i]);
    }
    printf("\n");

    hipFree(d_data);
    hipFree(d_data_cols);
    hipFree(d_rows);
    hipFree(d_V);
    hipFree(d_C);
}
