#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"



__global__ void count (char* str, char* key, int key_len, int* word_lengths, int num_words, int *n) {
    int idx = threadIdx.x;
    // printf("\n%d ---> %d", idx, idx);
    int start_idx = 0, length = word_lengths[idx];
    int flag = 1;
    for (int i=0; i<idx; i++) {
        start_idx += word_lengths[i] + 1;
    }
    if (key_len != length) {
        // printf("\n%d ---> Length Mismatch\n", idx);
        flag = 0;
    }
    

    // printf("%d ---> %d, %d\n", idx, start_idx, length);

    for (int i=start_idx; i<start_idx+length; i++) {
        if (str[i] != key[i-start_idx]) {
            // printf("\n%d ---> Char mismatch %c not in %c\n", idx, str[i], key[i-start_idx]);
            flag = 0;
            break;
        }
    }
    // printf("\n%d ---> %d\n", idx, flag);
    if (flag) {
        atomicAdd(n, 1);
        // printf("--->%d", *n);
    }
}

int main () {
    char str[100], key[10], *d_str, *d_key, ch;
    int n=0, *d_n;
    int word_lengths[50], num_words=1, *d_word_lengths, len_key;

    printf("Enter the words: ");
    fgets(str, 100, stdin);

    printf("Enter the key: ");
    scanf(" %s", key);

    int i=0, j=0;
    while (1) {
        ch = str[i++];
        // printf("ch ---> %c", ch);
        if (ch == '\0') {
            word_lengths[num_words-1] = j;
            break;
        }
        if (ch == ' ') {
            word_lengths[num_words-1] = j;
            j = 0;
            num_words++;
            continue;
        }
        if (ch >= 'a' && ch <= 'z' || ch >= 'A' && ch <= 'Z')
            j++;
    }

    len_key = strlen(key);

    /*
    printf("%s\n", str);
    printf("%s\n", key);
    printf("Num: %d\n", num_words);
    printf("Num_Key: %d\n", len_key);
    for (int i=0; i<num_words; i++) {
        printf("%d ", word_lengths[i]);
    }*/

    hipMalloc((void **) &d_str, strlen(str) * sizeof(char));
    hipMalloc((void **) &d_key, strlen(key) * sizeof(char));
    hipMalloc((void **) &d_word_lengths, num_words * sizeof(int));
    hipMalloc((void **) &d_n, 1 * sizeof(int));

    hipMemcpy(d_str, str, strlen(str) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, strlen(key) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word_lengths, word_lengths, num_words * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_n, &n, 1 * sizeof(int), hipMemcpyHostToDevice);

    count <<< 1, num_words >>> (d_str, d_key, len_key, d_word_lengths, num_words, d_n);

    hipMemcpy(&n, d_n, sizeof(int), hipMemcpyDeviceToHost);

    printf("Frequency of occurrence of key in the sentence: %d\n", n);

    hipFree(d_str);
    hipFree(d_key);
    hipFree(d_word_lengths);
    hipFree(d_n);
}
