#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"



__global__ void change (char* str, char* res, int len_str) {
    int idx = threadIdx.x;
    int startIdx = 0, endIdx;
    for (int i=0; i<idx; i++) 
        startIdx += len_str - i;
    endIdx = startIdx + len_str - idx;
    for (int i=startIdx; i<endIdx; i++) {
        res[i] = str[i - startIdx];
    }
}

int main () {
    char str[20], res[100];
    char *d_str, *d_res;

    printf("Enter the words: ");
    fgets(str, 20, stdin);

    int len_str = strlen(str), len_res=0;
    for (int i=1; i<=len_str; i++)
        len_res += i;
    res[len_res] = '\0';

    printf("%s\n", str);
    printf("Num_str: %d\n", len_str);
    printf("Num_res: %d\n", len_res);

    hipMalloc((void **) &d_str, len_str * sizeof(char));
    hipMalloc((void **) &d_res, len_res * sizeof(char));

    hipMemcpy(d_str, str, len_str * sizeof(char), hipMemcpyHostToDevice);

    change <<< 1, len_str >>> (d_str, d_res, len_str);

    hipMemcpy(res, d_res, len_res * sizeof(char), hipMemcpyDeviceToHost);

    printf("\nResulatant string: %s\n", res);

    hipFree(d_str);
    hipFree(d_res);
}
