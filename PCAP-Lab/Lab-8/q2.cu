#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"



__global__ void matMulRow (int* A, int* B, int* C, int wa, int wb) {
    int rowId = threadIdx.x, val;
    for (int colId=0; colId<wb; colId++) {
        val = 0;
        for (int k=0; k<wa; k++) {
            val += A[rowId * wa + k] * B[k * wb + colId];
        }
        C[rowId * wb + colId] = val;
    }
}

__global__ void matMulCol (int* A, int* B, int* C, int ha, int wa) {
    int colId = threadIdx.x, val;
    int wb = blockDim.x;
    for (int rowId=0; rowId<ha; rowId++) {
        val = 0;
        for (int k=0; k<wa; k++) {
            val += A[rowId * wa + k] * B[k * wb + colId];
        }
        C[rowId * wb + colId] = val;
    }
}

__global__ void matMulElement (int* A, int* B, int* C, int wa) {
    int rowId = threadIdx.x, colId = threadIdx.y;
    int wb = blockDim.y;
    int val = 0;
    for (int k=0; k<wa; k++) {
        val += A[rowId * wa + k] * B[k * wb + colId];
    }
    C[rowId * wb + colId] = val;
}

__host__ void displayMatrix (int *mat, int h, int w) {
    for (int i=0; i<h; i++) {
        for (int j=0; j<w; j++) {
            printf("%d ", mat[i * w + j]);
        }
        printf("\n");
    }
}

int main () {
    int ha, wa, hb, wb;

    printf("Enter number of rows of matrix A: ");
    scanf(" %d", &ha);
    printf("Enter number of columns of matrix A: ");
    scanf(" %d", &wa);
    printf("Enter number of rows of matrix B: ");
    scanf(" %d", &hb);
    printf("Enter number of columns of matrix B: ");
    scanf(" %d", &wb);

    int A[ha*wa], B[hb*wb], C[ha*wb], *d_A, *d_B, *d_C;

    printf("Enter %d elements of matrix A:\n", ha*wa);

    for (int i=0; i<ha*wa; i++) {
        scanf(" %d", A+i);
    }

    printf("Enter %d elements of matrix B:\n", hb*wb);

    for (int i=0; i<hb*wb; i++) {
        scanf(" %d", B+i);
    }

    hipMalloc((void **) &d_A, ha * wa * sizeof(int));
    hipMalloc((void **) &d_B, hb * wb * sizeof(int));
    hipMalloc((void **) &d_C, ha * wb * sizeof(int));

    hipMemcpy(d_A, A, ha * wa * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, hb * wb * sizeof(int), hipMemcpyHostToDevice);

    int x = -1;
    while (1) {
        printf("\n1: Row-wise \n2: Column-wise \n3: Element-wise \n0: Exit \nEnter method to use for multiplication: ");
        scanf(" %d", &x);
        if (x == 0) {
            printf("Exiting...\n");
            break;
        }

        if (x == 1)
            matMulRow <<< 1, ha >>> (d_A, d_B, d_C, wa, wb);
        else if (x == 2)
            matMulCol <<< 1, wb >>> (d_A, d_B, d_C, ha, wa);
        else {
            dim3 blockDim(ha, wb);
            matMulElement <<< 1, blockDim >>> (d_A, d_B, d_C, wa);
        }
            
        hipMemcpy(C, d_C, ha * wb * sizeof(int), hipMemcpyDeviceToHost);
    
        printf("\nResultant matrix C:\n");
        displayMatrix(C, ha, wb);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
