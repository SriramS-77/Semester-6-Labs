#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"



__global__ void matAddRow (int* A, int* B, int* C, int wa) {
    int rowId = threadIdx.x;
    for (int colId=0; colId<wa; colId++) {
        C[rowId * wa + colId] = A[rowId * wa + colId] + B[rowId * wa + colId];
    }
}

__global__ void matAddCol (int* A, int* B, int* C, int ha) {
    int colId = threadIdx.x, wa = blockDim.x;
    for (int rowId=0; rowId<ha; rowId++) {
        C[rowId * wa + colId] = A[rowId * wa + colId] + B[rowId * wa + colId];
    }
}

__global__ void matAddElement (int* A, int* B, int* C) {
    int rowId = threadIdx.x, colId = threadIdx.y, wa = blockDim.y;
    C[rowId * wa + colId] = A[rowId * wa + colId] + B[rowId * wa + colId];
}

__host__ void displayMatrix (int *mat, int h, int w) {
    for (int i=0; i<h; i++) {
        for (int j=0; j<w; j++) {
            printf("%d ", mat[i * w + j]);
        }
        printf("\n");
    }
}

int main () {
    int ha, wa;

    printf("Enter number of rows of matrix A: ");
    scanf(" %d", &ha);
    printf("Enter number of columns of matrix A: ");
    scanf(" %d", &wa);

    int A[ha*wa], B[ha*wa], C[ha*wa], *d_A, *d_B, *d_C;

    printf("Enter %d elements of matrix A:\n", ha*wa);

    for (int i=0; i<ha*wa; i++) {
        scanf(" %d", A+i);
    }

    printf("Enter %d elements of matrix B:\n", ha*wa);

    for (int i=0; i<ha*wa; i++) {
        scanf(" %d", B+i);
    }

    hipMalloc((void **) &d_A, ha * wa * sizeof(int));
    hipMalloc((void **) &d_B, ha * wa * sizeof(int));
    hipMalloc((void **) &d_C, ha * wa * sizeof(int));

    hipMemcpy(d_A, A, ha * wa * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, ha * wa * sizeof(int), hipMemcpyHostToDevice);

    int x = -1;
    while (1) {
        printf("\n1: Row-wise \n2: Column-wise \n3: Element-wise \n0: Exit \nEnter method to use for addition: ");
        scanf(" %d", &x);
        if (x == 0) {
            printf("Exiting...\n");
            break;
        }

        if (x == 1)
            matAddRow <<< 1, ha >>> (d_A, d_B, d_C, wa);
        else if (x == 2)
            matAddCol <<< 1, wa >>> (d_A, d_B, d_C, ha);
        else {
            dim3 blockDim(ha, wa);
            matAddElement <<< 1, blockDim >>> (d_A, d_B, d_C);
        }
            
        hipMemcpy(C, d_C, ha * wa * sizeof(int), hipMemcpyDeviceToHost);
    
        printf("\nResultant matrix C:\n");
        displayMatrix(C, ha, wa);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
