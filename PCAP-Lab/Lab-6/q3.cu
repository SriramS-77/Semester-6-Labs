#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"



__global__ void odd_even (int* arr, int n) {
    int idx = threadIdx.x;
    int temp, cycles=n/2;

    if (n & 1)
        cycles++;

    for (int i=0; i<cycles; i++) {
        if (idx % 2 == 0 && idx < n-1) {
            if (arr[idx] > arr[idx+1]) {
                temp = arr[idx];
                arr[idx] = arr[idx+1];
                arr[idx+1] = temp;
            }
        }
        else if (idx < n-1) {
            if (arr[idx] > arr[idx+1]) {
                temp = arr[idx];
                arr[idx] = arr[idx+1];
                arr[idx+1] = temp;
            }
        }
        __syncthreads();
    }
}

int main () {
    int *arr;
    int *d_arr;
    int n;

    printf("Enter length of array: ");
    scanf(" %d", &n);
    arr = (int*) calloc(n, sizeof(int));

    printf("Enter elements of array:\n");
    for (int i=0; i<n; i++) {
        scanf(" %d", arr+i);
    }

    int n_bytes = n * sizeof(int);

    hipMalloc((void **) &d_arr, n_bytes);

    hipMemcpy(d_arr, arr, n_bytes, hipMemcpyHostToDevice);

    odd_even <<< 1, n >>> (d_arr, n);

    hipMemcpy(arr, d_arr, n_bytes, hipMemcpyDeviceToHost);

    printf("Resultant array after sorting:\n");
    for (int i=0; i<n; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
}
