#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"



__global__ void conv (int* arr, int *mask, int *res, int width, int mask_width) {
    int index = threadIdx.x;
    int total_padding = mask_width - 1;
    int left_padding = total_padding / 2;
    // int right_padding = total_padding - left_padding;
    int sum = 0;

    for (int i=index; i<index+mask_width; i++) {
        if (i < left_padding || i - left_padding >= width) {
            continue;
        }
        sum += arr[i-left_padding] * mask[i-index];
    }
    // printf("%d ---> %d\n", index, sum);
    res[index] = sum;
}

int main () {
    int *arr, *mask, *res;
    int *d_arr, *d_mask, *d_res;
    int width, mask_width;

    printf("Enter length of array: ");
    scanf(" %d", &width);
    arr = (int*) calloc(width, sizeof(int));
    res = (int*) calloc(width, sizeof(int));
    printf("Enter elements of array:\n");
    for (int i=0; i<width; i++) {
        scanf(" %d", arr+i);
    }
    printf("Enter length of mask: ");
    scanf(" %d", &mask_width);
    mask = (int*) calloc(mask_width, sizeof(int));
    printf("Enter elements of mask:\n");
    for (int i=0; i<mask_width; i++) {
        scanf(" %d", mask+i);
    }

    int width_bytes = width * sizeof(int);
    int mask_width_bytes = mask_width * sizeof(int);

    hipMalloc((void **) &d_arr, width_bytes);
    hipMalloc((void **) &d_mask, mask_width_bytes);
    hipMalloc((void **) &d_res, width_bytes);

    hipMemcpy(d_arr, arr, width_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, mask_width_bytes, hipMemcpyHostToDevice);

    conv <<< 1, width >>> (d_arr, d_mask, d_res, width, mask_width);

    hipMemcpy(res, d_res, width_bytes, hipMemcpyDeviceToHost);

    printf("Resultant array after convolution:\n");
    for (int i=0; i<width; i++) {
        printf("%d ", res[i]);
    }
    printf("\n");

    hipFree(d_arr);
    hipFree(d_mask);
    hipFree(d_res);
}
