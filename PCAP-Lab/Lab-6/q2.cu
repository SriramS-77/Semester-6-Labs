#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"



__global__ void sort (int* arr, int n) {
    int idx = threadIdx.x;
    int pos = 0, temp;

    for (int i=1; i<idx; i++) {
        if (arr[i] > arr[pos])
            pos = i;
    }
    temp = arr[pos];
    arr[pos] = arr[idx];
    arr[idx] = temp;

    __syncthreads();
}

int main () {
    int *arr;
    int *d_arr;
    int n;

    printf("Enter length of array: ");
    scanf(" %d", &n);
    arr = (int*) calloc(n, sizeof(int));

    printf("Enter elements of array:\n");
    for (int i=0; i<n; i++) {
        scanf(" %d", arr+i);
    }

    int n_bytes = n * sizeof(int);

    hipMalloc((void **) &d_arr, n_bytes);

    hipMemcpy(d_arr, arr, n_bytes, hipMemcpyHostToDevice);

    sort <<< 1, n >>> (d_arr, n);

    hipMemcpy(arr, d_arr, n_bytes, hipMemcpyDeviceToHost);

    printf("Resultant array after sorting:\n");
    for (int i=0; i<n; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
}
