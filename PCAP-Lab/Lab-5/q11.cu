#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void add (int *d_a, int *d_b, int *d_c, int n) {
    int idx = threadIdx.x;
    d_c[idx] = d_a[idx] + d_b[idx];
}

int main () {
    int *a, *b, *c, n;
    int *d_a, *d_b, *d_c;   

    printf("Enter size of arrays: ");
    scanf(" %d", &n);

    a = (int*) calloc(n, sizeof(int));
    b = (int*) calloc(n, sizeof(int));
    c = (int*) calloc(n, sizeof(int));

    printf("Enter elements of array 1:\n");
    for (int i=0; i<n; i++) {
        scanf(" %d", a + i);
    }
    printf("Enter elements of array 2:\n");
    for (int i=0; i<n; i++) {
        scanf(" %d", b + i);
    }

    int size = sizeof(int) * n;

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 blockSize(n, 1, 1);

    add <<< 1, blockSize >>> (d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i=0; i<n; i++) {
        printf("%d ", c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
