#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void sin (float *d_a, float *d_b, int n) {
    int idx = threadIdx.x;
    d_b[idx] = sinf(d_a[idx]);
}

int main () {
    float *a, *b;
    int n;
    float *d_a, *d_b;

    printf("Enter size of arrays: ");
    scanf(" %d", &n);

    a = (float*) calloc(n, sizeof(float));
    b = (float*) calloc(n, sizeof(float));

    printf("Enter elements of 1D vector:\n");
    for (int i=0; i<n; i++) {
        scanf(" %f", a + i);
    }

    int size = sizeof(float) * n;

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 gridSize(1, 1, 1);
    dim3 blockSize(n, 1, 1);

    sin <<< gridSize, blockSize >>> (d_a, d_b, n);
    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i=0; i<n; i++) {
        printf("%f ", b[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
}
